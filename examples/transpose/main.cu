#include "transpose.cuh"
#include "transpose.h"
#include "cuda_utils.cuh"
#include <iostream>
#include <numeric>

#define uint size_t

template <typename T> void fill_matrix(T *A, uint m, uint n) {
    std::iota(A, A + m * n, 0);
}

template <typename T> void executeCUDA(T *A, T *cuda_A, size_t N) {
    int *d_A = nullptr, *d_out_A = nullptr;
    uint nbytes = sizeof(T) * N * N;

    CUDACheck(hipMalloc((void **)&d_A, nbytes));
    CUDACheck(hipMalloc((void **)&d_out_A, nbytes));

    CUDACheck(hipMemcpy(d_A, A, nbytes, hipMemcpyHostToDevice));

    uint threads = N > 1024 ? 1024 : N;
    uint blocks = N > 1024 ? N / 1024 + 1 : 1;
    transpose_parallel_per_element<<<blocks, threads>>>(d_A, d_out_A, N, 32);

    CUDACheck(hipMemcpy(cuda_A, d_out_A, nbytes, hipMemcpyDeviceToHost));
    hipFree(d_A);
    hipFree(d_out_A);
}

template <typename T> void executeISPC(T *A, T *ispc_A, uint N) {
    int threads = N > 1024 ? 1024 : N;
    int blocks = N > 1024 ? N / 1024 + 1 : 1;
    ispc::Dim3 grid{blocks, 1, 1};
    ispc::Dim3 block{threads, 1, 1};
    ispc::transpose_parallel_per_element(grid, block, 0, A, ispc_A, N, 32);
}

template <typename T> bool compareResults(T *A, T *B, uint N) {
    bool result = false;
    for (uint i = 0; i < N; i++) {
        for (uint j = 0; j < N; j++) {
            if (A[i * N + j] != B[i * N + j]) {
                std::cerr << "error at (i, j)=(" << i << ", " << j << ")\n";
                result = true;
            }
        }
    }
    return result;
}

int main(int argc, char *argv[]) {
    size_t N = 32;
    if (argc == 2) {
        N = strtoul(argv[argc - 1], nullptr, 10);
    }
    int *A = new int[N * N], *cuda_A = new int[N * N], *ispc_A = new int[N * N];
    fill_matrix(A, N, N);

    executeCUDA(A, cuda_A, N);
    executeISPC(A, ispc_A, N);
    if(compareResults(cuda_A, ispc_A, N))
        return 1;
    return 0;
}