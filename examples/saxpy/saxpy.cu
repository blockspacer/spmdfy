#include <hip/hip_runtime.h>

__global__ void saxpy(float *x, float *y, int n, float a) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

